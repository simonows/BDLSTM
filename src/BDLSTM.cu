#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include "BDLSTM.cuh"


double[] SigmoidA(double[] x)
{
    double[] result = new double[x.GetLength(0)];
    for (int i=0; i < x.GetLength(0); i++)
    {
        result[i] = 1.0/(1.0 + Math.Exp(-x[i]));
    }
    return result;
}


double[] matadd1D(double[] m1, double[] m2)
{
    double[] result = new double[m1.GetLength(0)];
    for (int i = 0; i < m1.GetLength(0); i++)
    {
        result[i] = m1[i] + m2[i];
    }
    return result;
}


double[] matmult2(double[]arr1,double[,]arr2)
{
    double temp = 0;
    double[] result = new double[arr2.GetLength(1)];

    for (int i = 0; i < arr2.GetLength(1); i++)
    {
        for (int j = 0; j < arr1.GetLength(0); j++)
        {
            temp += arr1[j] * arr2[j, i];
        }
        result[i] = temp;
        temp = 0;
    }
    return result;
}


double[] mult(double[]arr1,double[]arr2)
{
    double[] result = new double[arr1.GetLength(0)];
    if(arr1.GetLength(0) == arr2.GetLength(0))
    {
        for(int i=0; i < arr1.GetLength(0); i++)
        {
            result[i] = arr1[i] * arr2[i];
        }
    }
    else
    {
        throw new Exception("columns not equal");
    }
    return result;
}


double[] Tanh(double[] arr)
{
    double[] result = new double[arr.GetLength(0)];

    for(int i=0; i<arr.Length; i++)
    {
        result[i] = Math.Tanh(arr[i]);
    }
    return result;
}


__host__ int BDLSTM_cuda(
    double const **const input
  , double const* nnFlat
  , double const* nnLong
  , double const* nnShort
){
    int rc = 0;

    double[] lstm_output = new double[256];
    double[] f_lstm_output = new double[256];
    double[] b_lstm_output = new double[256];
    double[] f_lstm_state = new double[256];
    double[] b_lstm_state = new double[256];

    for (int i = 0; i < lstm_output.Length; i++)
    {
        lstm_output[i] = 0.001;
        f_lstm_output[i] = 0.001;
        b_lstm_output[i] = 0.001;
        f_lstm_state[i] = 0.001;
        b_lstm_state[i] = 0.001;
    }

    nnFlat = nnLong = nnShort = -0.01;

    for (int i = 0; i < _input.Length; i++)
    {
        double[] f_iiput = _input[i];
        double[] b_iiput = _input[_input.Length - i - 1];

        double[] f_input_gate = SigmoidA(matadd1D(matadd1D(matmult2(f_iiput,f_ig_2D),(matmult2(lstm_output,f_ih_2D))),f_bi));
        double[] b_input_gate = SigmoidA(matadd1D(matadd1D(matmult2(b_iiput,b_ig_2D),(matmult2(lstm_output,b_ih_2D))),b_bi));

        double[] f_forget_gate = SigmoidA(matadd1D(matadd1D(matmult2(f_iiput,f_fg_2D),(matmult2(lstm_output,f_fh_2D))),f_bf));
        double[] b_forget_gate = SigmoidA(matadd1D(matadd1D(matmult2(b_iiput,b_fg_2D),(matmult2(lstm_output,b_fh_2D))),b_bf));

        double[] f_output_gate = SigmoidA(matadd1D(matadd1D(matmult2(f_iiput,f_og_2D),(matmult2(lstm_output,f_oh_2D))),f_bo));
        double[] b_output_gate = SigmoidA(matadd1D(matadd1D(matmult2(b_iiput,b_og_2D),(matmult2(lstm_output,b_oh_2D))),f_bo));

        double[] f_memory_cell = Tanh(matadd1D(matadd1D(matmult2(f_iiput,f_mc_2D),(matmult2(lstm_output,f_mh_2D))),f_bm));
        double[] b_memory_cell = Tanh(matadd1D(matadd1D(matmult2(b_iiput,b_mc_2D),(matmult2(lstm_output,b_mh_2D))),b_bm));

        f_lstm_state  = matadd1D((mult(f_lstm_state,f_input_gate)),(mult(f_forget_gate, f_memory_cell)));
        b_lstm_state  = matadd1D((mult(b_lstm_state,b_input_gate)),(mult(b_forget_gate, b_memory_cell)));

        f_lstm_output = mult(f_output_gate,(Tanh(f_lstm_state)));
        b_lstm_output = mult(f_output_gate,(Tanh(b_lstm_state)));

        lstm_output = Tanh(matadd1D(f_lstm_output,b_lstm_output));
    }

    double[] model = matadd1D(matmult2(lstm_output, wo_2D), bol);

    double[] nnModel = Softmax(model);

    nnFlat = nnModel[0];
    nnLong = nnModel[1];
    nnShort= nnModel[2];


    return rc;
}

__global__ int GPU_LSTM()
{

}

